#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <thrust/transform.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cub/device/device_reduce.cuh>
#include <cub/iterator/transform_input_iterator.cuh>

#include <gtsam_points/types/point_cloud_gpu.hpp>
#include <gtsam_points/types/gaussian_voxelmap_cpu.hpp>
#include <gtsam_points/types/gaussian_voxelmap_gpu.hpp>
#include <gtsam_points/cuda/check_error.cuh>
#include <gtsam_points/cuda/kernels/vector3_hash.cuh>
#include <gtsam_points/cuda/cuda_malloc_async.hpp>

namespace gtsam_points {

namespace {

struct transform_means_kernel {
  transform_means_kernel(const thrust::device_ptr<const Eigen::Isometry3f>& transform_ptr) : transform_ptr(transform_ptr) {}

  __device__ Eigen::Vector3f operator()(const Eigen::Vector3f& x) const {
    const Eigen::Isometry3f& transform = *thrust::raw_pointer_cast(transform_ptr);
    return transform.linear() * x + transform.translation();
  }

  const thrust::device_ptr<const Eigen::Isometry3f> transform_ptr;
};

struct transform_covs_kernel {
  transform_covs_kernel(const thrust::device_ptr<const Eigen::Isometry3f>& transform_ptr) : transform_ptr(transform_ptr) {}

  __device__ Eigen::Matrix3f operator()(const Eigen::Matrix3f& cov) const {
    const Eigen::Isometry3f& transform = *thrust::raw_pointer_cast(transform_ptr);
    return transform.linear() * cov * transform.linear().transpose();
  }

  const thrust::device_ptr<const Eigen::Isometry3f> transform_ptr;
};
}  // namespace

PointCloud::Ptr merge_frames_gpu(
  const std::vector<Eigen::Isometry3d>& poses,
  const std::vector<PointCloud::ConstPtr>& frames,
  double downsample_resolution,
  ihipStream_t* stream) {
  //
  int num_all_points = 0;
  std::vector<Eigen::Isometry3f> h_poses(poses.size());
  for (int i = 0; i < poses.size(); i++) {
    h_poses[i] = poses[i].cast<float>();
    num_all_points += frames[i]->size();
  }

  Eigen::Isometry3f* d_poses;
  check_error << hipMallocAsync(&d_poses, sizeof(Eigen::Isometry3f) * poses.size(), stream);
  check_error << hipMemcpyAsync(d_poses, h_poses.data(), sizeof(Eigen::Isometry3f) * poses.size(), hipMemcpyHostToDevice, stream);

  Eigen::Vector3f* all_points;
  Eigen::Matrix3f* all_covs;
  check_error << hipMallocAsync(&all_points, sizeof(Eigen::Vector3f) * num_all_points, stream);
  check_error << hipMallocAsync(&all_covs, sizeof(Eigen::Matrix3f) * num_all_points, stream);

  const thrust::device_ptr<Eigen::Vector3f> all_points_ptr(all_points);
  const thrust::device_ptr<Eigen::Matrix3f> all_covs_ptr(all_covs);

  size_t begin = 0;
  for (int i = 0; i < frames.size(); i++) {
    const auto& frame = frames[i];
    const thrust::device_ptr<const Eigen::Isometry3f> transform_ptr(d_poses + i);
    const thrust::device_ptr<const Eigen::Vector3f> points_ptr(frame->points_gpu);
    const thrust::device_ptr<const Eigen::Matrix3f> covs_ptr(frame->covs_gpu);

    thrust::transform(
      thrust::cuda::par_nosync.on(stream),
      points_ptr,
      points_ptr + frame->size(),
      all_points_ptr + begin,
      transform_means_kernel(transform_ptr));
    thrust::transform(thrust::cuda::par.on(stream), covs_ptr, covs_ptr + frame->size(), all_covs_ptr + begin, transform_covs_kernel(transform_ptr));
    begin += frame->size();
  }

  check_error << hipStreamSynchronize(stream);

  PointCloud all_frames;
  all_frames.num_points = num_all_points;
  all_frames.points_gpu = all_points;
  all_frames.covs_gpu = all_covs;

  GaussianVoxelMapGPU downsampling(downsample_resolution, num_all_points, 10, 1e-3, stream);
  downsampling.insert(all_frames);

  const int num_voxels = downsampling.voxelmap_info.num_voxels;
  const Eigen::Vector3f* voxel_means = downsampling.voxel_means;
  const Eigen::Matrix3f* voxel_covs = downsampling.voxel_covs;

  std::vector<Eigen::Vector3f> means(num_voxels);
  std::vector<Eigen::Matrix3f> covs(num_voxels);

  check_error << hipMemcpyAsync(means.data(), voxel_means, sizeof(Eigen::Vector3f) * num_voxels, hipMemcpyDeviceToHost, stream);
  check_error << hipMemcpyAsync(covs.data(), voxel_covs, sizeof(Eigen::Matrix3f) * num_voxels, hipMemcpyDeviceToHost, stream);
  check_error << hipStreamSynchronize(stream);

  check_error << hipFreeAsync(d_poses, stream);
  check_error << hipFreeAsync(all_points, stream);
  check_error << hipFreeAsync(all_covs, stream);

  auto merged = std::make_shared<PointCloudGPU>();
  merged->add_points(means, stream);
  merged->add_covs(covs, stream);

  return merged;
}

namespace {
// point coord -> voxel coord conversion
struct overlap_count_kernel {
public:
  overlap_count_kernel(const GaussianVoxelMapGPU& voxelmap, const thrust::device_ptr<const Eigen::Isometry3f>& delta_ptr)
  : voxelmap_info_ptr(voxelmap.voxelmap_info_ptr),
    buckets_ptr(voxelmap.buckets),
    delta_ptr(delta_ptr) {}

  __host__ __device__ bool operator()(const Eigen::Vector3f& x) const {
    const auto& info = *thrust::raw_pointer_cast(voxelmap_info_ptr);
    const auto& trans = *thrust::raw_pointer_cast(delta_ptr);

    Eigen::Vector3f x_ = trans.linear() * x + trans.translation();
    int voxel_index = lookup_voxel(info.max_bucket_scan_count, info.num_buckets, thrust::raw_pointer_cast(buckets_ptr), info.voxel_resolution, x_);

    return voxel_index >= 0;
  }

  thrust::device_ptr<const VoxelMapInfo> voxelmap_info_ptr;
  thrust::device_ptr<const VoxelBucket> buckets_ptr;

  thrust::device_ptr<const Eigen::Isometry3f> delta_ptr;
};

struct bool_or_kernel {
  __host__ __device__ bool operator()(const thrust::tuple<bool, bool>& b) const { return thrust::get<0>(b) || thrust::get<1>(b); }
};

template <typename T_DST>
struct cast_kernel {
  template <typename T_SRC>
  __host__ __device__ T_DST operator()(const T_SRC& value) const {
    return static_cast<T_DST>(value);
  }
};
}  // namespace

double
overlap_gpu(const GaussianVoxelMap::ConstPtr& target_, const PointCloud::ConstPtr& source, const Eigen::Isometry3f* delta_gpu, ihipStream_t* stream) {
  if (!source->check_points_gpu()) {
    std::cerr << "error: GPU source points have not been allocated!!" << std::endl;
    abort();
  }

  auto target = std::dynamic_pointer_cast<const GaussianVoxelMapGPU>(target_);
  if (!target) {
    std::cerr << "error: Failed to cast target voxelmap to GaussianVoxelMapGPU!!" << std::endl;
    abort();
  }

  bool* overlap;
  check_error << hipMallocAsync(&overlap, sizeof(bool) * source->size(), stream);
  thrust::device_ptr<bool> overlap_ptr(overlap);

  thrust::transform(
    thrust::cuda::par_nosync.on(stream),
    thrust::device_ptr<Eigen::Vector3f>(source->points_gpu),
    thrust::device_ptr<Eigen::Vector3f>(source->points_gpu) + source->size(),
    overlap_ptr,
    overlap_count_kernel(*target, thrust::device_ptr<const Eigen::Isometry3f>(delta_gpu)));

  int* num_inliers;
  check_error << hipMallocAsync(&num_inliers, sizeof(int), stream);

  char* temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, overlap, num_inliers, source->size(), thrust::plus<int>(), 0, stream);
  check_error << hipMallocAsync(&temp_storage, temp_storage_bytes, stream);
  hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, overlap, num_inliers, source->size(), thrust::plus<int>(), 0, stream);

  int num_inliers_cpu = 0;
  check_error << hipMemcpyAsync(&num_inliers_cpu, num_inliers, sizeof(int), hipMemcpyDeviceToHost, stream);
  check_error << hipFreeAsync(overlap, stream);
  check_error << hipFreeAsync(temp_storage, stream);
  check_error << hipFreeAsync(num_inliers, stream);
  check_error << hipStreamSynchronize(stream);

  return static_cast<double>(num_inliers_cpu) / source->size();
}

double
overlap_gpu(const GaussianVoxelMap::ConstPtr& target_, const PointCloud::ConstPtr& source, const Eigen::Isometry3d& delta, ihipStream_t* stream) {
  if (!source->points_gpu) {
    std::cerr << "error: GPU source points have not been allocated!!" << std::endl;
    abort();
  }

  auto target = std::dynamic_pointer_cast<const GaussianVoxelMapGPU>(target_);
  if (!target) {
    std::cerr << "error: Failed to cast target voxelmap to GaussianVoxelMapGPU!!" << std::endl;
    abort();
  }

  Eigen::Isometry3f h_delta = delta.cast<float>();
  Eigen::Isometry3f* d_delta;
  check_error << hipMallocAsync(&d_delta, sizeof(Eigen::Isometry3f), stream);
  check_error << hipMemcpyAsync(d_delta, h_delta.data(), sizeof(Eigen::Isometry3f), hipMemcpyHostToDevice, stream);

  double overlap = overlap_gpu(target, source, d_delta, stream);
  check_error << hipFreeAsync(d_delta, stream);

  return overlap;
}

double overlap_gpu(
  const std::vector<GaussianVoxelMap::ConstPtr>& targets_,
  const PointCloud::ConstPtr& source,
  const std::vector<Eigen::Isometry3d>& deltas_,
  ihipStream_t* stream) {
  if (!source->points_gpu) {
    std::cerr << "error: GPU source points have not been allocated!!" << std::endl;
    abort();
  }

  std::vector<GaussianVoxelMapGPU::ConstPtr> targets(targets_.size());
  for (int i = 0; i < targets_.size(); i++) {
    targets[i] = std::dynamic_pointer_cast<const GaussianVoxelMapGPU>(targets_[i]);
    if (!targets[i]) {
      std::cerr << "error: Failed to cast target voxelmap to GaussianVoxelMapGPU!!" << std::endl;
    }
  }

  std::vector<Eigen::Isometry3f> h_deltas(deltas_.size());
  std::transform(deltas_.begin(), deltas_.end(), h_deltas.begin(), [](const Eigen::Isometry3d& delta) { return delta.cast<float>(); });

  Eigen::Isometry3f* deltas;
  check_error << hipMallocAsync(&deltas, sizeof(Eigen::Isometry3f) * h_deltas.size(), stream);
  check_error << hipMemcpyAsync(deltas, h_deltas.data(), sizeof(Eigen::Isometry3f) * h_deltas.size(), hipMemcpyHostToDevice, stream);
  thrust::device_ptr<Eigen::Isometry3f> deltas_ptr(deltas);

  bool* overlap;
  check_error << hipMallocAsync(&overlap, sizeof(bool) * source->size(), stream);
  check_error << hipMemsetAsync(overlap, 0, sizeof(bool) * source->size(), stream);
  thrust::device_ptr<bool> overlap_ptr(overlap);

  for (int i = 0; i < targets.size(); i++) {
    overlap_count_kernel overlap_kernel(*targets[i], deltas_ptr + i);
    auto first = thrust::make_transform_iterator(thrust::device_ptr<Eigen::Vector3f>(source->points_gpu), overlap_kernel);
    auto last = thrust::make_transform_iterator(thrust::device_ptr<Eigen::Vector3f>(source->points_gpu) + source->size(), overlap_kernel);

    thrust::transform(
      thrust::cuda::par_nosync.on(stream),
      thrust::make_zip_iterator(thrust::make_tuple(overlap_ptr, first)),
      thrust::make_zip_iterator(thrust::make_tuple(overlap_ptr + source->size(), last)),
      overlap_ptr,
      bool_or_kernel());
  }

  int* num_inliers;
  check_error << hipMallocAsync(&num_inliers, sizeof(int), stream);

  void* temp_storage = nullptr;
  size_t temp_storage_bytes = 0;

  hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, overlap, num_inliers, source->size(), thrust::plus<int>(), 0, stream);
  check_error << hipMallocAsync(&temp_storage, temp_storage_bytes, stream);
  hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, overlap, num_inliers, source->size(), thrust::plus<int>(), 0, stream);

  int num_inliers_cpu;
  check_error << hipMemcpyAsync(&num_inliers_cpu, num_inliers, sizeof(int), hipMemcpyDeviceToHost, stream);
  check_error << hipFreeAsync(deltas, stream);
  check_error << hipFreeAsync(overlap, stream);
  check_error << hipFreeAsync(temp_storage, stream);
  check_error << hipFreeAsync(num_inliers, stream);

  return static_cast<double>(num_inliers_cpu) / source->size();
}

std::vector<double> overlap_gpu(
  const std::vector<GaussianVoxelMap::ConstPtr>& targets_,
  const std::vector<PointCloud::ConstPtr>& sources,
  const std::vector<Eigen::Isometry3d>& Ts_target_source_,
  ihipStream_t* stream) {
  if (targets_.size() != sources.size()) {
    std::cerr << "error: The number of target voxelmaps and source point clouds must be the same!!" << std::endl;
    abort();
  }

  size_t max_num_points = 0;

  std::vector<GaussianVoxelMapGPU::ConstPtr> targets(targets_.size());
  for (int i = 0; i < targets_.size(); i++) {
    targets[i] = std::dynamic_pointer_cast<const GaussianVoxelMapGPU>(targets_[i]);
    if (!targets[i]) {
      std::cerr << "error: Failed to cast target voxelmap to GaussianVoxelMapGPU!!" << std::endl;
    }

    if (!sources[i]->has_points_gpu()) {
      std::cerr << "error: GPU source points have not been allocated!!" << std::endl;
    }

    max_num_points = std::max(max_num_points, sources[i]->size());
  }

  std::vector<Eigen::Isometry3f> h_deltas(Ts_target_source_.size());
  std::transform(Ts_target_source_.begin(), Ts_target_source_.end(), h_deltas.begin(), [](const Eigen::Isometry3d& delta) {
    return delta.cast<float>();
  });

  Eigen::Isometry3f* deltas;
  check_error << hipMallocAsync(&deltas, sizeof(Eigen::Isometry3f) * Ts_target_source_.size(), stream);
  check_error << hipMemcpyAsync(deltas, h_deltas.data(), sizeof(Eigen::Isometry3f) * Ts_target_source_.size(), hipMemcpyHostToDevice, stream);

  bool* overlap;
  check_error << hipMallocAsync(&overlap, sizeof(bool) * max_num_points, stream);

  int* num_inliers;
  check_error << hipMallocAsync(&num_inliers, sizeof(float) * sources.size(), stream);

  char* temp_storage = nullptr;
  size_t temp_storage_bytes = 0;

  for (int i = 0; i < targets_.size(); i++) {
    const auto& source = sources[i];
    const auto& target = targets[i];
    thrust::transform(
      thrust::cuda::par_nosync.on(stream),
      thrust::device_ptr<Eigen::Vector3f>(source->points_gpu),
      thrust::device_ptr<Eigen::Vector3f>(source->points_gpu) + source->size(),
      thrust::device_ptr<bool>(overlap),
      overlap_count_kernel(*target, thrust::device_ptr<const Eigen::Isometry3f>(deltas + i)));

    size_t required_temp_storage_bytes = 0;
    hipcub::DeviceReduce::Reduce(nullptr, required_temp_storage_bytes, overlap, num_inliers + i, source->size(), thrust::plus<int>(), 0, stream);

    if (temp_storage_bytes < required_temp_storage_bytes) {
      temp_storage_bytes = required_temp_storage_bytes;
      check_error << hipFreeAsync(temp_storage, stream);
      check_error << hipMallocAsync(&temp_storage, temp_storage_bytes, stream);
    }

    hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, overlap, num_inliers + i, source->size(), thrust::plus<int>(), 0, stream);
  }

  std::vector<int> h_num_inliers(sources.size());
  check_error << hipMemcpyAsync(h_num_inliers.data(), num_inliers, sizeof(int) * sources.size(), hipMemcpyDeviceToHost, stream);

  std::vector<double> overlaps(sources.size());
  for (int i = 0; i < sources.size(); i++) {
    overlaps[i] = static_cast<double>(h_num_inliers[i]) / sources[i]->size();
  }

  check_error << hipFreeAsync(deltas, stream);
  check_error << hipFreeAsync(overlap, stream);
  check_error << hipFreeAsync(temp_storage, stream);
  check_error << hipFreeAsync(num_inliers, stream);

  return overlaps;
}

}  // namespace gtsam_points