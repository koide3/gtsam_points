// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/factors/integrated_vgicp_derivatives.cuh>

#include <iostream>

#include <gtsam_points/cuda/check_error.cuh>
#include <gtsam_points/cuda/kernels/linearized_system.cuh>
#include <gtsam_points/cuda/kernels/vgicp_derivatives.cuh>
#include <gtsam_points/cuda/stream_temp_buffer_roundrobin.hpp>
#include <gtsam_points/cuda/cuda_malloc_async.hpp>

#include <gtsam_points/types/point_cloud_gpu.hpp>
#include <gtsam_points/types/gaussian_voxelmap_gpu.hpp>

namespace gtsam_points {

IntegratedVGICPDerivatives::IntegratedVGICPDerivatives(
  const GaussianVoxelMapGPU::ConstPtr& target,
  const PointCloud::ConstPtr& source,
  ihipStream_t* ext_stream,
  std::shared_ptr<TempBufferManager> temp_buffer)
: enable_offloading(false),
  enable_surface_validation(false),
  inlier_update_thresh_trans(1e-6),
  inlier_update_thresh_angle(1e-6),
  target(target),
  source(source),
  external_stream(true),
  stream(ext_stream),
  temp_buffer(temp_buffer),
  num_inliers(0),
  source_inliers(nullptr) {
  //
  if (stream == nullptr) {
    external_stream = false;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  }

  if (this->temp_buffer == nullptr) {
    this->temp_buffer.reset(new TempBufferManager());
  }

  check_error << hipMallocAsync(&num_inliers_gpu, sizeof(int), stream);
  // check_error << hipHostRegister(&num_inliers, sizeof(int), hipHostRegisterDefault);
}

IntegratedVGICPDerivatives::~IntegratedVGICPDerivatives() {
  check_error << hipFreeAsync(source_inliers, stream);
  check_error << hipFreeAsync(num_inliers_gpu, stream);
  // check_error << hipHostUnregister(&num_inliers);

  if (!external_stream) {
    hipStreamDestroy(stream);
  }
}

void IntegratedVGICPDerivatives::sync_stream() {
  check_error << hipStreamSynchronize(stream);
}

void IntegratedVGICPDerivatives::touch_points() {
  if (!enable_offloading) {
    return;
  }

  auto target_ = const_cast<GaussianVoxelMapGPU*>(target.get());
  target_->touch(stream);

  auto source_gpu_const = dynamic_cast<const PointCloudGPU*>(source.get());
  if (!source_gpu_const) {
    return;
  }

  auto source_gpu = const_cast<PointCloudGPU*>(source_gpu_const);
  source_gpu->touch(stream);
}

LinearizedSystem6 IntegratedVGICPDerivatives::linearize(const Eigen::Isometry3f& x) {
  thrust::device_vector<Eigen::Isometry3f> x_ptr(1);
  thrust::device_vector<LinearizedSystem6> output_ptr(1);

  x_ptr[0] = x;

  reset_inliers(x, thrust::raw_pointer_cast(x_ptr.data()));
  issue_linearize(thrust::raw_pointer_cast(x_ptr.data()), thrust::raw_pointer_cast(output_ptr.data()));
  sync_stream();

  LinearizedSystem6 linearized = output_ptr[0];

  return linearized;
}

double IntegratedVGICPDerivatives::compute_error(const Eigen::Isometry3f& d_xl, const Eigen::Isometry3f& d_xe) {
  thrust::device_vector<Eigen::Isometry3f> xs_ptr(2);
  xs_ptr[0] = d_xl;
  xs_ptr[1] = d_xe;
  thrust::device_vector<float> output_ptr(1);

  issue_compute_error(
    thrust::raw_pointer_cast(xs_ptr.data()),
    thrust::raw_pointer_cast(xs_ptr.data() + 1),
    thrust::raw_pointer_cast(output_ptr.data()));
  sync_stream();

  float error = output_ptr[0];
  return error;
}

size_t IntegratedVGICPDerivatives::memory_usage_gpu() const {
  return sizeof(int) + sizeof(int) * num_inliers;
}

bool IntegratedVGICPDerivatives::loaded_on_gpu() const {
  return source_inliers;
}

bool IntegratedVGICPDerivatives::offload_gpu(ihipStream_t* stream) {
  if (!source_inliers) {
    return false;
  }

  check_error << hipFreeAsync(source_inliers, stream);
  source_inliers = nullptr;
  return true;
}

bool IntegratedVGICPDerivatives::reload_gpu(ihipStream_t* stream) {
  if (source_inliers) {
    return false;
  }
  return false;
}

}  // namespace gtsam_points